#include "hip/hip_runtime.h"
#include <kernel.h>

// Thread block size
#define THREADS_BS 16

// Device multiplication function called by Mul() 
// Compute C = A * B 
//	wA is the width of A 
//	wB is the width of B
__global__ void matrixMult_cuda(REAL* C, REAL* A,  REAL * B, int wA, int wB)
{
   // Block and thread index 
   int bx = blockIdx.x;
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   // Index of the first/last sub-matrix of A processed by the block 
   int aBegin = wA * THREADS_BS * by;
   int aEnd   = aBegin + wA - 1;

   // Step size used to iterate through the sub-matrices of A 
   int aStep = THREADS_BS;
   // Index of the first sub-matrix of B processed by the block 
   int bBegin = THREADS_BS * bx;
   // Step size used to iterate through the sub-matrices of B 
   int bStep = THREADS_BS * wB;

   // The element of the block sub-matrix that is computed  by the thread 
   REAL Csub = 0;

   // Loop over all sub-matrices (A & B) required to compute the block sub-matrix 
   for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {

      // Shared memory for sub-matrix of A & B
      __shared__ REAL As[THREADS_BS][THREADS_BS];
      __shared__ REAL Bs[THREADS_BS][THREADS_BS];

      // Load the matrices from global memory to shared memory;  
      // each thread loads one element of each matrix 
      As[ty][tx] = A[a + wA * ty + tx];  
      Bs[ty][tx] = B[b + wB * ty + tx];

      // Synchronize to make sure the matrices are loaded 
      __syncthreads();

      // Multiply the two matrices together; each thread computes one element 
      // of the block sub-matrix  
      for (int k = 0; k < THREADS_BS; ++k)
         Csub += As[ty][k] * Bs[k][tx];

      // Synchronize to make sure that the preceding computation is done before
      // loading two new sub-matrices of A and B in the next iteration
      __syncthreads();
   }

   // Write the block sub-matrix to global memory; each thread writes one element
   int c = wB * THREADS_BS * by + THREADS_BS * bx;
   C[c + wB*ty + tx] += Csub;
}
